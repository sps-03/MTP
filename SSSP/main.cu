#include <vector>
#include <map>
#include <set>
#include <algorithm>
#include <chrono>
#include <utility>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define MAX_INT 2147483647

// structure for storing edge information
struct edgeInfo {
    int src, dest, weight;
};

// structure for update information
struct updateInfo {
    char type;
    int src, dest, weight;
};

// HOST FUNCTIONS:
bool compareTwoEdges(const edgeInfo &a, const edgeInfo &b);
bool compareTwoEdgesR(const edgeInfo &a, const edgeInfo &b);
void printArray(int *arr, int len);
void checkCudaError();
unsigned long long SSSP_GPU(int numVertices, int numEdges, int *csrOffsets, int *csrCords, int *csrWeights, 
                            int *distances, int *parent, int source);
void processUpdates(int numVertices, int numEdges, int *distances, int *parent, updateInfo *&updates, 
                    int batchSize, int *csrOffsets, int *csrCords, int *csrWeights, 
                    int *csrOffsetsR, int *csrCordsR, int *csrWeightsR, 
                    int &numEdgesDiffCsr, int *&diffCsrOffsets, int *&diffCsrCords, int *&diffCsrWeights, 
                    int &numEdgesDiffCsrR, int *&diffCsrOffsetsR, int *&diffCsrCordsR, int *&diffCsrWeightsR);

// DEVICE FUNCTIONS:
template <typename T>
__global__ void init_kernel(T *array, T val, int arraySize);
__global__ void sssp_kernel(int *csrOffsets_d, int *csrCords_d, int *csrWeights_d, int *distances_d, 
                            int *parent_d, int *locks_d, int numVertices, bool *modified_d, 
                            bool *modified_next_d, bool *finished_d);
__global__ void delete_edges(int batchSize, int *distances_d, int *parent_d, updateInfo *updates_d, 
                             bool *modifiedD_d, int *csrOffsets_d, int *csrCords_d, int *csrWeights_d,
                             int *csrOffsetsR_d, int *csrCordsR_d, int *csrWeightsR_d,
                             int *diffCsrOffsets_d, int *diffCsrCords_d, int *diffCsrWeights_d,
                             int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, int *diffCsrWeightsR_d);
__global__ void mark_descendants(int *distances_d, int *parent_d, bool *modifiedD_d, 
                                 int numVertices, bool *finished_d);
__global__ void mark_not_reachable(int *distances_d, int *parent_d, int numVertices, bool *modifiedD_d);
__global__ void fetch_and_update(int *csrOffsetsR_d, int *csrCordsR_d, int *csrWeightsR_d, 
                                 int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, int *diffCsrWeightsR_d, 
                                 int *distances_d, int *parent_d, int numVertices, 
                                 bool *modifiedD_d, bool *finished_d);
__global__ void add_edges_csr(int batchSize, int *distances_d, int *parent_d, updateInfo *updates_d, 
                              bool *modifiedA_d, int *csrOffsets_d, int *csrCords_d, int *csrWeights_d,
                              int *csrOffsetsR_d, int *csrCordsR_d, int *csrWeightsR_d,
                              int *diffCsrOffsets_next_d, int *diffCsrOffsetsR_next_d);
__global__ void add_diff_edge_counts(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                     int *diffCsrWeights_d, int *diffCsrOffsetsR_d, int *diffCsrCordsR_d,
                                     int *diffCsrWeightsR_d, int *diffCsrOffsets_next_d, 
                                     int *diffCsrOffsetsR_next_d);
template <typename T>
__global__ void copy(T *destArr, T *srcArr, int arraySize);
__global__ void prefix_sum(int numVertices, int off, int *diffCsrOffsets_next_d, 
                           int *diffCsrOffsetsR_next_d, int *tempArr_d, int *tempArrR_d);
__global__ void copy_edges_diffcsr(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                   int *diffCsrWeights_d, int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, 
                                   int *diffCsrWeightsR_d, int *diffCsrOffsets_next_d, 
                                   int *diffCsrCords_next_d, int *diffCsrWeights_next_d,
                                   int *diffCsrOffsetsR_next_d, int *diffCsrCordsR_next_d, 
                                   int *diffCsrWeightsR_next_d);
__global__ void add_edges_diffcsr(int batchSize, int *distances_d, int *parent_d, 
                                  updateInfo *updates_d, bool *modifiedA_d, int *diffCsrOffsets_d, 
                                  int *diffCsrCords_d, int *diffCsrWeights_d, int *diffCsrOffsetsR_d, 
                                  int *diffCsrCordsR_d, int *diffCsrWeightsR_d);
__global__ void push_and_update(int *csrOffsets_d, int *csrCords_d, int *csrWeights_d, 
                                int *distances_d, int *parent_d, int *diffCsrOffsets_d, 
                                int *diffCsrCords_d, int *diffCsrWeights_d, int *locks_d, 
                                int numVertices, bool *modifiedA_d, bool *finished_d);


// main function
// usage: ./a.out inputFile updateFile [percentUpdate] [isDirected]
int main(int argc, char **argv) {
    // if input or update file names are not provided then exit
    if(argc < 3) {
        printf("Enter the input and update file path in the command line.\n");
        return 0;
    }

    // read file names
    char *inputFile = argv[1];
    char *updateFile = argv[2];
    
    // open input and update files
    FILE *inputFilePtr = fopen(inputFile, "r");
    FILE *updateFilePtr = fopen(updateFile, "r");
    
    // if not able to open the input file then exit
    if(inputFilePtr == NULL) {
        printf("Failed to open the input file.\n");
        return 0;
    }

    // if not able to open the update file then exit
    if(updateFilePtr == NULL) {
        printf("Failed to open the update file.\n");
        return 0;
    }

    // declaration of variables
    int numVertices, numEdges, startVertex;
    fscanf(inputFilePtr, "%d", &numVertices);
    fscanf(inputFilePtr, "%d", &numEdges);
    // fscanf(inputFilePtr, "%d", &startVertex);
    startVertex = 0;

    // to store the input graph in COO format
    std::vector<edgeInfo> COO(numEdges);

    // read from the input file and populate the COO
    for(int i=0; i<numEdges; i++) {
        int src, dest, weight;
        fscanf(inputFilePtr, "%d %d", &src, &dest);
        weight = 1;

        COO[i].src = src;
        COO[i].dest = dest;
        COO[i].weight = weight;
    }

    // close input file
    fclose(inputFilePtr);

    // sort the COO
    std::sort(COO.begin(), COO.end(), compareTwoEdges);

    // converting the graph in COO format to CSR format
    int *csrOffsets = (int*)malloc(sizeof(int)*(numVertices+1));
    int *csrCords = (int*)malloc(sizeof(int)*(numEdges));
    int *csrWeights = (int*)malloc(sizeof(int)*(numEdges));

    // initialize the Offsets array
    for(int i=0; i<=numVertices; i++) csrOffsets[i] = 0;

    // update the Coordinates and Weights array
    for(int i=0; i<numEdges; i++) {
        csrCords[i] = COO[i].dest;
        csrWeights[i] = COO[i].weight;
    }

    // update the Offsets array
    for(int i=0; i<numEdges; i++) csrOffsets[COO[i].src+1]++;		    // store the frequency
    for(int i=0; i<numVertices; i++) csrOffsets[i+1] += csrOffsets[i];	// do cumulative sum

    // sort the COO (for reverseCSR)
    std::sort(COO.begin(), COO.end(), compareTwoEdgesR);

    // converting the graph in COO format to reverseCSR format
    int *csrOffsetsR = (int*)malloc(sizeof(int)*(numVertices+1));
    int *csrCordsR = (int*)malloc(sizeof(int)*(numEdges));
    int *csrWeightsR = (int*)malloc(sizeof(int)*(numEdges));

    // initialize the Offsets array
    for(int i=0; i<=numVertices; i++) csrOffsetsR[i] = 0;

    // update the Coordinates and Weights array
    for(int i=0; i<numEdges; i++) {
        csrCordsR[i] = COO[i].src;
        csrWeightsR[i] = COO[i].weight;
    }

    // update the Offsets array
    for(int i=0; i<numEdges; i++) csrOffsetsR[COO[i].dest+1]++;		        // store the frequency
    for(int i=0; i<numVertices; i++) csrOffsetsR[i+1] += csrOffsetsR[i];	// do cumulative sum

    std::vector<edgeInfo>().swap(COO);

    // converting the graph to CSRs done

    // get graph type
    bool isDirected = true;
    if(argc >= 5 && atoi(argv[4])==0) isDirected = false;

    // shortest distances from start vertex
    int *distances_gpu = (int*)malloc(sizeof(int)*numVertices);

    // parent array
    int *parent = (int*)malloc(sizeof(int)*numVertices);

    // compute the shortest paths
    unsigned long long gpuTotalPathSum = SSSP_GPU(numVertices, numEdges, csrOffsets, csrCords, csrWeights, 
                                         distances_gpu, parent, startVertex);
    printf("Initial graph: Dist=%llu\n", gpuTotalPathSum);

    // will be using Diff-CSR along with CSR for dynamic graph
    int numEdgesDiffCsr = 0;
    int numEdgesDiffCsrR = 0;
    int *diffCsrOffsets, *diffCsrCords, *diffCsrWeights;
    int *diffCsrOffsetsR, *diffCsrCordsR, *diffCsrWeightsR;
    diffCsrOffsets = (int*)malloc(sizeof(int)*(numVertices+1));
    diffCsrOffsetsR = (int*)malloc(sizeof(int)*(numVertices+1));
    diffCsrCords = NULL; diffCsrWeights = NULL;
    diffCsrCordsR = NULL; diffCsrWeightsR = NULL;
    for(int i=0; i<=numVertices; i++) {
        diffCsrOffsets[i] = 0;
        diffCsrOffsetsR[i] = 0;
    }
    
    // for measuring time for each update
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

    // start updates
    int numUpdates, batchSize, percentUpdate;
    char type;
    int u, v, w;
    double totalTime = 0.0;
    
    percentUpdate = 10;
    numUpdates = percentUpdate * 0.01 * numEdges;
    if(argc >= 4) {
        percentUpdate = atoi(argv[3]);
        numUpdates = percentUpdate * 0.01 * numEdges;
    }
    if(!isDirected && numUpdates&1) numUpdates++;
    batchSize = numUpdates;
    updateInfo *updates = (updateInfo*) malloc(sizeof(updateInfo) * batchSize);
    for(int i=0; i < numUpdates; i+=batchSize) {
        for(int j=0; j<batchSize; j++) {
            fscanf(updateFilePtr, " %c", &type);
            fscanf(updateFilePtr, "%d", &u);
            fscanf(updateFilePtr, "%d", &v);
            w = 1;
            
            updates[j] = {type, u, v, w};
            if(!isDirected) updates[++j] = {type, v, u, w};
        }

        // call the function to process the updates for current batch
        start = std::chrono::high_resolution_clock::now();
        processUpdates(numVertices, numEdges, distances_gpu, parent, updates, batchSize,
                       csrOffsets, csrCords, csrWeights, csrOffsetsR, csrCordsR, csrWeightsR, 
                       numEdgesDiffCsr, diffCsrOffsets, diffCsrCords, diffCsrWeights, 
                       numEdgesDiffCsrR, diffCsrOffsetsR, diffCsrCordsR, diffCsrWeightsR);
        end = std::chrono::high_resolution_clock::now();

        // for measuring total time taken for updates
        std::chrono::duration<double, std::milli> timeTaken = end-start;
        totalTime += timeTaken.count();
        printf("Total time taken for %d percent updates: %.3f ms\n", percentUpdate, totalTime);
    }
    
    gpuTotalPathSum = 0;
    for(int i=0; i<numVertices; i++) {
        if(distances_gpu[i] != MAX_INT)
            gpuTotalPathSum += distances_gpu[i];
    }
    printf("Final graph: Dist=%llu\n\n", gpuTotalPathSum);

    // free memory allocated on host
    free(csrOffsets);
    free(csrCords);
    free(csrWeights);
    free(csrOffsetsR);
    free(csrCordsR);
    free(csrWeightsR);
    free(diffCsrOffsets);
    free(diffCsrCords);
    free(diffCsrWeights);
    free(diffCsrOffsetsR);
    free(diffCsrCordsR);
    free(diffCsrWeightsR);
    free(distances_gpu);
    free(parent);
    free(updates);

    // close files
    fclose(updateFilePtr);

    return 0;
}

// comparator function
bool compareTwoEdges(const edgeInfo &a, const edgeInfo &b) {
    if(a.src != b.src) return a.src < b.src;
    return a.dest < b.dest;
}

// comparator function for reverse CSR
bool compareTwoEdgesR(const edgeInfo &a, const edgeInfo &b) {
    if(a.dest != b.dest) return a.dest < b.dest;
    return a.src < b.src;
}

// host function to print an array content
void printArray(int *arr, int len) {
    for(int i=0; i<len; i++) {
        printf("%d ", arr[i]);
    } printf("\n");
}

// check for cudaError
void checkCudaError() {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}

// host function for parallel bellman ford (fixed point)
unsigned long long SSSP_GPU(int numVertices, int numEdges, int *csrOffsets, int *csrCords, int *csrWeights, 
                            int *distances, int *parent, int source=0) {
    // launch config
    const int numThreads = 1024;
    const int numBlocksV = (numVertices+numThreads-1)/numThreads;
    // const numBlocksE = (numEdges+numOfThreads-1)/numThreads;

    // pointers for arrays on host
    bool *modified = (bool*)malloc(sizeof(bool)*numVertices);
    bool *finished = (bool*)malloc(sizeof(bool));
    
    // pointers for arrays on device
    int *csrOffsets_d, *csrCords_d, *csrWeights_d;
    int *distances_d, *parent_d, *locks_d;
    bool *modified_d, *modified_next_d, *finished_d;

    // allocate memory on device
    hipMalloc(&csrOffsets_d, sizeof(int)*(numVertices+1));
    hipMalloc(&csrCords_d, sizeof(int)*(numEdges));
    hipMalloc(&csrWeights_d, sizeof(int)*(numEdges));
    hipMalloc(&distances_d, sizeof(int)*numVertices);
    hipMalloc(&parent_d, sizeof(int)*numVertices);
    hipMalloc(&locks_d, sizeof(int)*numVertices);
    hipMalloc(&modified_d, sizeof(bool)*numVertices);
    hipMalloc(&modified_next_d, sizeof(bool)*numVertices);
    hipMalloc(&finished_d, sizeof(bool));

    // initialize the host arrays
    for(int i=0; i<numVertices; i++) {
        distances[i] = MAX_INT;
        parent[i] = -1;
        modified[i] = false;
    }
    distances[source] = 0;
    modified[source] = true;
    *finished = false;

    // for recording the total time taken
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy to device
    hipMemcpy(csrOffsets_d, csrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(csrCords_d, csrCords, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(csrWeights_d, csrWeights, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(distances_d, distances, sizeof(int)*(numVertices), hipMemcpyHostToDevice);
    hipMemcpy(parent_d, parent, sizeof(int)*(numVertices), hipMemcpyHostToDevice);
    hipMemcpy(modified_d, modified, sizeof(bool)*(numVertices), hipMemcpyHostToDevice);

    // call kernel to compute edge relaxing till no more updates or at max "numVertices-1" times
    int iter = 0;
    init_kernel<bool><<<numBlocksV, numThreads>>>(modified_next_d, false, numVertices);
    init_kernel<int><<<numBlocksV, numThreads>>>(locks_d, 0, numVertices);
    while(*finished != true) {
        init_kernel<bool><<<1, 1>>>(finished_d, true, 1);
        sssp_kernel<<<numBlocksV, numThreads>>>(csrOffsets_d, csrCords_d, csrWeights_d, distances_d, parent_d, 
                                                locks_d, numVertices, modified_d, modified_next_d, finished_d);
        init_kernel<bool><<<numBlocksV, numThreads>>>(modified_d, false, numVertices);
        hipMemcpy(finished, finished_d, sizeof(bool), hipMemcpyDeviceToHost);

        bool *tempPtr = modified_next_d;
        modified_next_d = modified_d;
        modified_d = tempPtr;

        if(++iter >= numVertices-1) break;
    }

    // check for error
    checkCudaError();
    
    // copy distances back to host
    hipMemcpy(distances, distances_d, sizeof(int)*(numVertices), hipMemcpyDeviceToHost);

    // copy parent array back to host
    hipMemcpy(parent, parent_d, sizeof(int)*(numVertices), hipMemcpyDeviceToHost);

    // print time taken
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Static Graph: SSSP using GPU \nTime Taken: %.6f ms \nIterations: %d\n", milliseconds, iter);

    // free up the memory
    free(modified);
    free(finished);
    hipFree(csrOffsets_d);
    hipFree(csrCords_d);
    hipFree(csrWeights_d);
    hipFree(distances_d);
    hipFree(parent_d);
    hipFree(locks_d);
    hipFree(modified_d);
    hipFree(modified_next_d);
    hipFree(finished_d);

    unsigned long long sum = 0;
    for(int i=0; i<numVertices; i++) {
        if(distances[i] != MAX_INT)
            sum += distances[i];
    }
    return sum;
}

// process updates (in batches)
void processUpdates(int numVertices, int numEdges, int *distances, int *parent, updateInfo *&updates, 
                    int batchSize, int *csrOffsets, int *csrCords, int *csrWeights, 
                    int *csrOffsetsR, int *csrCordsR, int *csrWeightsR, 
                    int &numEdgesDiffCsr, int *&diffCsrOffsets, int *&diffCsrCords, int *&diffCsrWeights, 
                    int &numEdgesDiffCsrR, int *&diffCsrOffsetsR, int *&diffCsrCordsR, int *&diffCsrWeightsR) {
    // launch config
    const int numThreads = 1024;
    const int numBlocksB = (batchSize+numThreads-1)/numThreads;
    const int numBlocksV = (numVertices+numThreads-1)/numThreads;

    // pointers for arrays on host
    bool *finished = (bool*)malloc(sizeof(bool));
    
    // pointers for arrays on device
    int *distances_d, *parent_d, *locks_d;
    updateInfo *updates_d;
    int *csrOffsets_d, *csrCords_d, *csrWeights_d;
    int *csrOffsetsR_d, *csrCordsR_d, *csrWeightsR_d;
    int *diffCsrOffsets_d, *diffCsrCords_d, *diffCsrWeights_d;
    int *diffCsrOffsetsR_d, *diffCsrCordsR_d, *diffCsrWeightsR_d;
    bool *modifiedD_d, *modifiedA_d, *finished_d;

    // allocate memory on device
    hipMalloc(&distances_d, sizeof(int)*numVertices);
    hipMalloc(&parent_d, sizeof(int)*numVertices);
    hipMalloc(&locks_d, sizeof(int)*numVertices);
    hipMalloc(&updates_d, sizeof(updateInfo)*batchSize);
    hipMalloc(&csrOffsets_d, sizeof(int)*(numVertices+1));
    hipMalloc(&csrCords_d, sizeof(int)*(numEdges));
    hipMalloc(&csrWeights_d, sizeof(int)*(numEdges));
    hipMalloc(&csrOffsetsR_d, sizeof(int)*(numVertices+1));
    hipMalloc(&csrCordsR_d, sizeof(int)*(numEdges));
    hipMalloc(&csrWeightsR_d, sizeof(int)*(numEdges));
    hipMalloc(&diffCsrOffsets_d, sizeof(int)*(numVertices+1));
    hipMalloc(&diffCsrCords_d, sizeof(int)*(numEdgesDiffCsr));
    hipMalloc(&diffCsrWeights_d, sizeof(int)*(numEdgesDiffCsr));
    hipMalloc(&diffCsrOffsetsR_d, sizeof(int)*(numVertices+1));
    hipMalloc(&diffCsrCordsR_d, sizeof(int)*(numEdgesDiffCsrR));
    hipMalloc(&diffCsrWeightsR_d, sizeof(int)*(numEdgesDiffCsrR));
    hipMalloc(&modifiedD_d, sizeof(bool)*numVertices);
    hipMalloc(&modifiedA_d, sizeof(bool)*numVertices);
    hipMalloc(&finished_d, sizeof(bool));

    // for recording the time taken
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy to device
    hipMemcpy(distances_d, distances, sizeof(int)*(numVertices), hipMemcpyHostToDevice);
    hipMemcpy(parent_d, parent, sizeof(int)*(numVertices), hipMemcpyHostToDevice);
    hipMemcpy(updates_d, updates, sizeof(updateInfo)*(batchSize), hipMemcpyHostToDevice);
    hipMemcpy(csrOffsets_d, csrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(csrCords_d, csrCords, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(csrWeights_d, csrWeights, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(csrOffsetsR_d, csrOffsetsR, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(csrCordsR_d, csrCordsR, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(csrWeightsR_d, csrWeightsR, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrOffsets_d, diffCsrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrCords_d, diffCsrCords, sizeof(int)*(numEdgesDiffCsr), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrWeights_d, diffCsrWeights, sizeof(int)*(numEdgesDiffCsr), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrOffsetsR_d, diffCsrOffsetsR, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrCordsR_d, diffCsrCordsR, sizeof(int)*(numEdgesDiffCsrR), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrWeightsR_d, diffCsrWeightsR, sizeof(int)*(numEdgesDiffCsrR), hipMemcpyHostToDevice);    
    init_kernel<bool><<<numBlocksV, numThreads>>>(modifiedD_d, false, numVertices);
    init_kernel<bool><<<numBlocksV, numThreads>>>(modifiedA_d, false, numVertices);

    // delete the edges from CSR & diffCSR and mark modified nodes
    delete_edges<<<numBlocksB, numThreads>>>(batchSize, distances_d, parent_d, updates_d, modifiedD_d,
                                             csrOffsets_d, csrCords_d, csrWeights_d,
                                             csrOffsetsR_d, csrCordsR_d, csrWeightsR_d,
                                             diffCsrOffsets_d, diffCsrCords_d, diffCsrWeights_d,
                                             diffCsrOffsetsR_d, diffCsrCordsR_d, diffCsrWeightsR_d);
    
    // mark the descendants of modified nodes as modified
    int iter = 0;
    *finished = false;
    while(*finished != true) {
        init_kernel<bool><<<1, 1>>>(finished_d, true, 1);
        mark_descendants<<<numBlocksV, numThreads>>>(distances_d, parent_d, modifiedD_d, 
                                                     numVertices, finished_d);
        hipMemcpy(finished, finished_d, sizeof(bool), hipMemcpyDeviceToHost);
        if(++iter >= numVertices-1) break;
    }

    // set the distance and parent of marked nodes (MAX_INT, -1)
    mark_not_reachable<<<numBlocksV, numThreads>>>(distances_d, parent_d, numVertices, modifiedD_d);
    
    // update the distances and parents (pull based approach)
    iter = 0;
    *finished = false;
    while(*finished != true) {
        init_kernel<bool><<<1, 1>>>(finished_d, true, 1);
        fetch_and_update<<<numBlocksV, numThreads>>>(csrOffsetsR_d, csrCordsR_d, csrWeightsR_d, 
                                                     diffCsrOffsetsR_d, diffCsrCordsR_d, diffCsrWeightsR_d, 
                                                     distances_d, parent_d, numVertices, modifiedD_d, finished_d);
        hipMemcpy(finished, finished_d, sizeof(bool), hipMemcpyDeviceToHost);
        if(++iter >= numVertices-1) break;
    }
    
    // new diffCSR
    int *diffCsrOffsets_next_d, *diffCsrCords_next_d, *diffCsrWeights_next_d;
    int *diffCsrOffsetsR_next_d, *diffCsrCordsR_next_d, *diffCsrWeightsR_next_d;
    hipMalloc(&diffCsrOffsets_next_d, sizeof(int)*(numVertices+1));
    hipMalloc(&diffCsrOffsetsR_next_d, sizeof(int)*(numVertices+1));
    init_kernel<int><<<numBlocksV+1, numThreads>>>(diffCsrOffsets_next_d, 0, numVertices+1);
    init_kernel<int><<<numBlocksV+1, numThreads>>>(diffCsrOffsetsR_next_d, 0, numVertices+1);
    
    // add edges to available space in CSR
    add_edges_csr<<<numBlocksB, numThreads>>>(batchSize, distances_d, parent_d, updates_d, modifiedA_d,
                                              csrOffsets_d, csrCords_d, csrWeights_d,
                                              csrOffsetsR_d, csrCordsR_d, csrWeightsR_d,
                                              diffCsrOffsets_next_d, diffCsrOffsetsR_next_d);
    
    // get the count of valid edges in old diffCSR (for computing offsets for new diffCSR)
    add_diff_edge_counts<<<numBlocksV, numThreads>>>(numVertices, diffCsrOffsets_d, diffCsrCords_d, diffCsrWeights_d,
                                                     diffCsrOffsetsR_d, diffCsrCordsR_d, diffCsrWeightsR_d,
                                                     diffCsrOffsets_next_d, diffCsrOffsetsR_next_d);

    // compute offsets
    int *tempArr_d, *tempArrR_d;
    hipMalloc(&tempArr_d, sizeof(int)*(numVertices+1));
    hipMalloc(&tempArrR_d, sizeof(int)*(numVertices+1));
    copy<int><<<numBlocksV+1, numThreads>>>(tempArr_d, diffCsrOffsets_next_d, numVertices+1);
    copy<int><<<numBlocksV+1, numThreads>>>(tempArrR_d, diffCsrOffsetsR_next_d, numVertices+1);
    
    for(int off=1; off<=numVertices; off*=2) {
        prefix_sum<<<numBlocksV+1, numThreads>>>(numVertices, off, diffCsrOffsets_next_d, 
                                                 diffCsrOffsetsR_next_d, tempArr_d, tempArrR_d);
        copy<int><<<numBlocksV+1, numThreads>>>(tempArr_d, diffCsrOffsets_next_d, numVertices+1);
        copy<int><<<numBlocksV+1, numThreads>>>(tempArrR_d, diffCsrOffsetsR_next_d, numVertices+1);
    }
    
    hipFree(tempArr_d);
    hipFree(tempArrR_d);
    
    // populate new diffCSR
    hipMemcpy(&numEdgesDiffCsr, &diffCsrOffsets_next_d[numVertices], sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&numEdgesDiffCsrR, &diffCsrOffsetsR_next_d[numVertices], sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&diffCsrCords_next_d, sizeof(int)*(numEdgesDiffCsr));
    hipMalloc(&diffCsrWeights_next_d, sizeof(int)*(numEdgesDiffCsr));
    hipMalloc(&diffCsrCordsR_next_d, sizeof(int)*(numEdgesDiffCsrR));
    hipMalloc(&diffCsrWeightsR_next_d, sizeof(int)*(numEdgesDiffCsrR));
    init_kernel<int><<<((numEdgesDiffCsr)+numThreads)/numThreads, numThreads>>>(diffCsrWeights_next_d, MAX_INT, numEdgesDiffCsr);
    init_kernel<int><<<((numEdgesDiffCsrR)+numThreads)/numThreads, numThreads>>>(diffCsrWeightsR_next_d, MAX_INT, numEdgesDiffCsrR);
    
    // copy edges from previous diffCSR to new diffCSr
    copy_edges_diffcsr<<<numBlocksV, numThreads>>>(numVertices, diffCsrOffsets_d, diffCsrCords_d, diffCsrWeights_d, 
                                                   diffCsrOffsetsR_d, diffCsrCordsR_d, diffCsrWeightsR_d,
                                                   diffCsrOffsets_next_d, diffCsrCords_next_d, diffCsrWeights_next_d,
                                                   diffCsrOffsetsR_next_d, diffCsrCordsR_next_d, diffCsrWeightsR_next_d);

    hipFree(diffCsrOffsets_d); diffCsrOffsets_d = diffCsrOffsets_next_d;
    hipFree(diffCsrCords_d); diffCsrCords_d = diffCsrCords_next_d;
    hipFree(diffCsrWeights_d); diffCsrWeights_d = diffCsrWeights_next_d;
    hipFree(diffCsrOffsetsR_d); diffCsrOffsetsR_d = diffCsrOffsetsR_next_d;
    hipFree(diffCsrCordsR_d); diffCsrCordsR_d = diffCsrCordsR_next_d;
    hipFree(diffCsrWeightsR_d); diffCsrWeightsR_d = diffCsrWeightsR_next_d;

    // add remaining edges to new diffCSR
    add_edges_diffcsr<<<numBlocksB, numThreads>>>(batchSize, distances_d, parent_d, updates_d, modifiedA_d,
                                                  diffCsrOffsets_d, diffCsrCords_d, diffCsrWeights_d,
                                                  diffCsrOffsetsR_d, diffCsrCordsR_d, diffCsrWeightsR_d);

    // update the distances and parents (push based approach)
    *finished = false;
    iter = 0;
    init_kernel<int><<<numBlocksV, numThreads>>>(locks_d, 0, numVertices);
    while(*finished != true) {
        init_kernel<bool><<<1, 1>>>(finished_d, true, 1);
        push_and_update<<<numBlocksV, numThreads>>>(csrOffsets_d, csrCords_d, csrWeights_d, distances_d, parent_d, 
                                                    diffCsrOffsets_d, diffCsrCords_d, diffCsrWeights_d,
                                                    locks_d, numVertices, modifiedA_d, finished_d);
        hipMemcpy(finished, finished_d, sizeof(bool), hipMemcpyDeviceToHost);

        if(++iter >= numVertices-1) break;
    }
    
    // copy the arrays back to the host
    hipMemcpy(distances, distances_d, sizeof(int)*(numVertices), hipMemcpyDeviceToHost);
    hipMemcpy(parent, parent_d, sizeof(int)*(numVertices), hipMemcpyDeviceToHost);
    hipMemcpy(csrOffsets, csrOffsets_d, sizeof(int)*(numVertices+1), hipMemcpyDeviceToHost);
    hipMemcpy(csrCords, csrCords_d, sizeof(int)*(numEdges), hipMemcpyDeviceToHost);
    hipMemcpy(csrWeights, csrWeights_d, sizeof(int)*(numEdges), hipMemcpyDeviceToHost);
    hipMemcpy(csrOffsetsR, csrOffsetsR_d, sizeof(int)*(numVertices+1), hipMemcpyDeviceToHost);
    hipMemcpy(csrCordsR, csrCordsR_d, sizeof(int)*(numEdges), hipMemcpyDeviceToHost);
    hipMemcpy(csrWeightsR, csrWeightsR_d, sizeof(int)*(numEdges), hipMemcpyDeviceToHost);
    hipMemcpy(diffCsrOffsets, diffCsrOffsets_d, sizeof(int)*(numVertices+1), hipMemcpyDeviceToHost);
    diffCsrCords = (int*)realloc(diffCsrCords, sizeof(int)*(numEdgesDiffCsr));
    diffCsrWeights = (int*)realloc(diffCsrWeights, sizeof(int)*(numEdgesDiffCsr));
    hipMemcpy(diffCsrCords, diffCsrCords_d, sizeof(int)*(numEdgesDiffCsr), hipMemcpyDeviceToHost);
    hipMemcpy(diffCsrWeights, diffCsrWeights_d, sizeof(int)*(numEdgesDiffCsr), hipMemcpyDeviceToHost);
    hipMemcpy(diffCsrOffsetsR, diffCsrOffsetsR_d, sizeof(int)*(numVertices+1), hipMemcpyDeviceToHost);
    diffCsrCordsR = (int*)realloc(diffCsrCordsR, sizeof(int)*(numEdgesDiffCsrR));
    diffCsrWeightsR = (int*)realloc(diffCsrWeightsR, sizeof(int)*(numEdgesDiffCsrR));
    hipMemcpy(diffCsrCordsR, diffCsrCordsR_d, sizeof(int)*(numEdgesDiffCsrR), hipMemcpyDeviceToHost);
    hipMemcpy(diffCsrWeightsR, diffCsrWeightsR_d, sizeof(int)*(numEdgesDiffCsrR), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaError();
    
    // print the time taken
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time Taken: %.6f ms\n", milliseconds);

    // free up the memory
    free(finished);
    hipFree(distances_d);
    hipFree(parent_d);
    hipFree(locks_d);
    hipFree(updates_d);
    hipFree(csrOffsets_d);
    hipFree(csrCords_d);
    hipFree(csrWeights_d);
    hipFree(csrOffsetsR_d);
    hipFree(csrCordsR_d);
    hipFree(csrWeightsR_d);
    hipFree(diffCsrOffsets_d);
    hipFree(diffCsrCords_d);
    hipFree(diffCsrWeights_d);
    hipFree(diffCsrOffsetsR_d);
    hipFree(diffCsrCordsR_d);
    hipFree(diffCsrWeightsR_d);
    hipFree(modifiedD_d);
    hipFree(modifiedA_d);
    hipFree(finished_d);
}

// kernel for value initialization
template <typename T>
__global__ void init_kernel(T *array, T val, int arraySize) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < arraySize) array[id] = val;
}

// kernel for computing SSSP of static graph
__global__ void sssp_kernel(int *csrOffsets_d, int *csrCords_d, int *csrWeights_d, int *distances_d, 
                            int *parent_d, int *locks_d, int numVertices, bool *modified_d, 
                            bool *modified_next_d, bool *finished_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices && modified_d[id]==true && distances_d[id]!=MAX_INT) {
        int distToCurNode = distances_d[id];
        int v, newDist, lock;
        bool gotLock;
        for(int e=csrOffsets_d[id]; e<csrOffsets_d[id+1]; e++) {
            gotLock = false;
            v = csrCords_d[e];
            newDist = distToCurNode + csrWeights_d[e];
            do {
                if(gotLock==false) lock = atomicCAS(&locks_d[v], 0, 1);
                if(lock==0 && newDist < distances_d[v]) {
                    distances_d[v] = newDist;
                    parent_d[v] = id;
                    modified_next_d[v] = true;
                    *finished_d = false;
                }
                if(lock==0) {
                    gotLock = true;
                    lock = 1;
                    atomicExch(&locks_d[v], 0);
                }
            } while(gotLock == false);
        }
    }
}

// kernel for deleting edges from the graph (CSR and diffCSR)
__global__ void delete_edges(int batchSize, int *distances_d, int *parent_d, updateInfo *updates_d, 
                             bool *modifiedD_d, int *csrOffsets_d, int *csrCords_d, int *csrWeights_d,
                             int *csrOffsetsR_d, int *csrCordsR_d, int *csrWeightsR_d,
                             int *diffCsrOffsets_d, int *diffCsrCords_d, int *diffCsrWeights_d,
                             int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, int *diffCsrWeightsR_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type=='d') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        if(parent_d[v] >= 0 && parent_d[v]==u) {
            modifiedD_d[v] = true;
        }
        for(int i=csrOffsets_d[u]; i<csrOffsets_d[u+1]; i++) {
            if(csrCords_d[i]==v) {
                csrWeights_d[i] = MAX_INT;
                break;
            }
        }
        for(int i=csrOffsetsR_d[v]; i<csrOffsetsR_d[v+1]; i++) {
            if(csrCordsR_d[i]==u) {
                csrWeightsR_d[i] = MAX_INT;
                break;
            }
        }
        for(int i=diffCsrOffsets_d[u]; i<diffCsrOffsets_d[u+1]; i++) {
            if(diffCsrCords_d[i]==v) {
                diffCsrWeights_d[i] = MAX_INT;
                break;
            }
        }
        for(int i=diffCsrOffsetsR_d[v]; i<diffCsrOffsetsR_d[v+1]; i++) {
            if(diffCsrCordsR_d[i]==u) {
                diffCsrWeightsR_d[i] = MAX_INT;
                break;
            }
        }
    }
}

// kernel for marking all the descendants of modified nodes in SPT
__global__ void mark_descendants(int *distances_d, int *parent_d, bool *modifiedD_d, int numVertices,
                                 bool *finished_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices && modifiedD_d[id]==false) {
        int parent = parent_d[id];
        if(parent>=0 && modifiedD_d[parent]==true) {
            modifiedD_d[id] = true;
            *finished_d = false;
        }
    }
}

// kernel for setting distance and parent of marked nodes
__global__ void mark_not_reachable(int *distances_d, int *parent_d, int numVertices, bool *modifiedD_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices && modifiedD_d[id]==true) {
        distances_d[id] = MAX_INT;
        parent_d[id] = -1;
    }
}

// kernel for updating the distance and parent of marked nodes (as a result of deletions)
__global__ void fetch_and_update(int *csrOffsetsR_d, int *csrCordsR_d, int *csrWeightsR_d, 
                                 int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, int *diffCsrWeightsR_d, 
                                 int *distances_d, int *parent_d, int numVertices, 
                                 bool *modifiedD_d, bool *finished_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices && modifiedD_d[id]==true) {
        int u;
        for(int e=csrOffsetsR_d[id]; e<csrOffsetsR_d[id+1]; e++) {
            u = csrCordsR_d[e];
            if(distances_d[u] != MAX_INT && csrWeightsR_d[e] != MAX_INT) {
                if(distances_d[id] > distances_d[u]+csrWeightsR_d[e]) {
                    distances_d[id] = distances_d[u]+csrWeightsR_d[e];
                    parent_d[id] = u;
                    *finished_d = false;
                }
            }
        }
        for(int e=diffCsrOffsetsR_d[id]; e<diffCsrOffsetsR_d[id+1]; e++) {
            u = diffCsrCordsR_d[e];
            if(distances_d[u] != MAX_INT && diffCsrWeightsR_d[e] != MAX_INT) {
                if(distances_d[id] > distances_d[u]+diffCsrWeightsR_d[e]) {
                    distances_d[id] = distances_d[u]+diffCsrWeightsR_d[e];
                    parent_d[id] = u;
                    *finished_d = false;
                }
            }
        }
    }
}

// kernel for adding edges in to the CSR and marking modified nodes
__global__ void add_edges_csr(int batchSize, int *distances_d, int *parent_d, updateInfo *updates_d, 
                              bool *modifiedA_d, int *csrOffsets_d, int *csrCords_d, int *csrWeights_d,
                              int *csrOffsetsR_d, int *csrCordsR_d, int *csrWeightsR_d,
                              int *diffCsrOffsets_next_d, int *diffCsrOffsetsR_next_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type=='a') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        int w = updates_d[id].weight;
        int flag=0, flagR=0;
        if(distances_d[u]!=MAX_INT && distances_d[u]+w <= distances_d[v]) {
            modifiedA_d[u] = true;
        }
        for(int i=csrOffsets_d[u]; i<csrOffsets_d[u+1]; i++) {
            if(csrWeights_d[i]==MAX_INT) {
                flag = atomicCAS(&csrWeights_d[i], MAX_INT, w);
                if(flag==MAX_INT) {
                    updates_d[id].type='x';
                    csrCords_d[i] = v;
                    break;
                }
            }
        }
        for(int i=csrOffsetsR_d[v]; i<csrOffsetsR_d[v+1]; i++) {
            if(csrWeightsR_d[i]==MAX_INT) {
                flagR = atomicCAS(&csrWeightsR_d[i], MAX_INT, w);
                if(flagR==MAX_INT) {
                    if(updates_d[id].type=='a') updates_d[id].type='y';
                    else updates_d[id].type='z';
                    csrCordsR_d[i] = u;
                    break;
                }
            }
        }

        if(flag!=MAX_INT) atomicAdd(&diffCsrOffsets_next_d[u+1], 1);
        if(flagR!=MAX_INT) atomicAdd(&diffCsrOffsetsR_next_d[v+1], 1);
    }
}

// kernel for counting valid edges in diffCSR (for updating offset values)
__global__ void add_diff_edge_counts(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                     int *diffCsrWeights_d, int *diffCsrOffsetsR_d, int *diffCsrCordsR_d,
                                     int *diffCsrWeightsR_d, int *diffCsrOffsets_next_d, 
                                     int *diffCsrOffsetsR_next_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices) {
        for(int i=diffCsrOffsets_d[id]; i<diffCsrOffsets_d[id+1]; i++) {
            if(diffCsrWeights_d[i]!=MAX_INT) diffCsrOffsets_next_d[id+1] += 1;
        }
        for(int i=diffCsrOffsetsR_d[id]; i<diffCsrOffsetsR_d[id+1]; i++) {
            if(diffCsrWeightsR_d[i]!=MAX_INT) diffCsrOffsetsR_next_d[id+1] += 1;
        }
    }
}

// kernel for copying values from one array to another array
template <typename T>
__global__ void copy(T *destArr, T *srcArr, int arraySize) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < arraySize) destArr[id] = srcArr[id];
}

// kernel for prefix sum
__global__ void prefix_sum(int numVertices, int off, int *diffCsrOffsets_next_d, 
                           int *diffCsrOffsetsR_next_d, int *tempArr_d, int *tempArrR_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if((id <= numVertices) && (id >= off)) {
        diffCsrOffsets_next_d[id] += tempArr_d[id-off];
        diffCsrOffsetsR_next_d[id] += tempArrR_d[id-off];
    }
}

// kernel for copying the edges from old diffCSR to new diffCSR arrays
__global__ void copy_edges_diffcsr(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, int *diffCsrWeights_d, 
                                   int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, int *diffCsrWeightsR_d,
                                   int *diffCsrOffsets_next_d, int *diffCsrCords_next_d, int *diffCsrWeights_next_d,
                                   int *diffCsrOffsetsR_next_d, int *diffCsrCordsR_next_d, int *diffCsrWeightsR_next_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices) {
        int j=diffCsrOffsets_next_d[id];
        for(int i=diffCsrOffsets_d[id]; i<diffCsrOffsets_d[id+1]; i++) {
            if(diffCsrWeights_d[i]!=MAX_INT) {
                diffCsrCords_next_d[j] = diffCsrCords_d[i];
                diffCsrWeights_next_d[j] = diffCsrWeights_d[i];
                j++;
            }
        }
        j=diffCsrOffsetsR_next_d[id];
        for(int i=diffCsrOffsetsR_d[id]; i<diffCsrOffsetsR_d[id+1]; i++) {
            if(diffCsrWeightsR_d[i]!=MAX_INT) {
                diffCsrCordsR_next_d[j] = diffCsrCordsR_d[i];
                diffCsrWeightsR_next_d[j] = diffCsrWeightsR_d[i];
                j++;
            }
        }
    }
}

// kernel for adding remaining edges (from udpates) to diffCSR
__global__ void add_edges_diffcsr(int batchSize, int *distances_d, int *parent_d, updateInfo *updates_d, bool *modifiedA_d, 
                                  int *diffCsrOffsets_d, int *diffCsrCords_d, int *diffCsrWeights_d,
                                  int *diffCsrOffsetsR_d, int *diffCsrCordsR_d, int *diffCsrWeightsR_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && (updates_d[id].type=='a' || updates_d[id].type=='x' || updates_d[id].type=='y')) {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        int w = updates_d[id].weight;
        if(distances_d[u]!=MAX_INT && distances_d[u]+w <= distances_d[v]) {
            modifiedA_d[u] = true;
        }
        if(updates_d[id].type=='a' || updates_d[id].type=='y') {
            for(int i=diffCsrOffsets_d[u]; i<diffCsrOffsets_d[u+1]; i++) {
                if(diffCsrWeights_d[i]==MAX_INT) {
                    if(atomicCAS(&diffCsrWeights_d[i], MAX_INT, w)==MAX_INT) {
                        diffCsrCords_d[i] = v;
                        break;
                    }
                }
            }
        }
        if(updates_d[id].type=='a' || updates_d[id].type=='x') {
            for(int i=diffCsrOffsetsR_d[v]; i<diffCsrOffsetsR_d[v+1]; i++) {
                if(diffCsrWeightsR_d[i]==MAX_INT) {
                    if(atomicCAS(&diffCsrWeightsR_d[i], MAX_INT, w)==MAX_INT) {
                        diffCsrCordsR_d[i] = u;
                        break;
                    }
                }
            }
        }
    }
}

// kernel for updating the distance and parent of marked nodes (as a result of additions)
__global__ void push_and_update(int *csrOffsets_d, int *csrCords_d, int *csrWeights_d, int *distances_d, int *parent_d,
                                int *diffCsrOffsets_d, int *diffCsrCords_d, int *diffCsrWeights_d,
                                int *locks_d, int numVertices, bool *modifiedA_d, bool *finished_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices && modifiedA_d[id]==true && distances_d[id]!=MAX_INT) {
        int distToCurNode = distances_d[id];
        int v, newDist, lock;
        bool gotLock;
        for(int e=csrOffsets_d[id]; e<csrOffsets_d[id+1]; e++) {
            if(csrWeights_d[e] != MAX_INT) {
                gotLock = false;
                v = csrCords_d[e];
                newDist = distToCurNode + csrWeights_d[e];
                do {
                    if(gotLock==false) lock = atomicCAS(&locks_d[v], 0, 1);
                    if(lock==0 && newDist < distances_d[v]) {
                        distances_d[v] = newDist;
                        parent_d[v] = id;
                        modifiedA_d[v] = true;
                        *finished_d = false;
                    }
                    if(lock==0) {
                        gotLock = true;
                        lock = 1;
                        atomicExch(&locks_d[v], 0);
                    }
                } while(gotLock == false);
            }
        }
        for(int e=diffCsrOffsets_d[id]; e<diffCsrOffsets_d[id+1]; e++) {
            if(diffCsrWeights_d[e] != MAX_INT) {
                gotLock = false;
                v = diffCsrCords_d[e];
                newDist = distToCurNode + diffCsrWeights_d[e];
                do {
                    if(gotLock==false) lock = atomicCAS(&locks_d[v], 0, 1);
                    if(lock==0 && newDist < distances_d[v]) {
                        distances_d[v] = newDist;
                        parent_d[v] = id;
                        modifiedA_d[v] = true;
                        *finished_d = false;
                    }
                    if(lock==0) {
                        gotLock = true;
                        lock = 1;
                        atomicExch(&locks_d[v], 0);
                    }
                } while(gotLock == false);
            }
        }
    }
}