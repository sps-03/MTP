#include "../include/graph.cuh"

// host function for parallel bellman ford routine
long long SSSP_GPU(int numVertices, int numEdges, int *csrOffsets, int *csrCords, int *csrWeights, 
              int *distances, int source=0) {
    // initialize the distances array
    for(int i=0; i<numVertices; i++) distances[i] = INT_MAX>>1;
    distances[source] = 0;

    // launch config
    const int numThreads = 1024;
    const int numBlocksV = (numVertices+numThreads-1)/numThreads;
    // const numBlocksE = (numEdges+numOfThreads-1)/numThreads;

    // pointers for arrays on GPU
    int *csrOffsets_g, *csrCords_g, *csrWeights_g;
    int *distances_g;

    // allocate memory on GPU
    hipMalloc(&csrOffsets_g, sizeof(int)*(numVertices+1));
    hipMalloc(&csrCords_g, sizeof(int)*(numEdges));
    hipMalloc(&csrWeights_g, sizeof(int)*(numEdges));
    hipMalloc(&distances_g, sizeof(int)*numVertices);

    // copy to GPU
    hipMemcpy(csrOffsets_g, csrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(csrCords_g, csrCords, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(csrWeights_g, csrWeights, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(distances_g, distances, sizeof(int)*(numVertices), hipMemcpyHostToDevice);

    // call kernel to compute edge relaxing numVertices-1 times
    for(int i=0; i<numVertices-1; i++) {
      sssp_kernel<<<numBlocksV, numThreads>>>(csrOffsets_g, csrCords_g, csrWeights_g, distances_g, numVertices, numEdges);
      hipDeviceSynchronize();

      // check for error
      hipError_t error = hipGetLastError();
      if(error != hipSuccess) {
        // print the CUDA error message
        printf("CUDA error: %s\n", hipGetErrorString(error));
      }
    }

    // copy distances back to CPU
    hipMemcpy(distances, distances_g, sizeof(int)*(numVertices), hipMemcpyDeviceToHost);
    
    /////////////////////////////////////////////////////////////////////////////////
    // for(int i=0; i<=numVertices; i++) printf("%d ", csrOffsets[i]);
    // printf("\n");
    // for(int i=0; i<numEdges; i++) printf("%d ", csrCords[i]);
    // printf("\n");
    // for(int i=0; i<numEdges; i++) printf("%d ", csrWeights[i]);
    // printf("\n");
    /////////////////////////////////////////////////////////////////////////////////

    long long sum = 0;
    for(int i=0; i<numVertices; i++) sum += distances[i];
    return sum;
}