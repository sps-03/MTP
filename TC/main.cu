#include <vector>
#include <algorithm>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX_INT 2147483647

// structure for storing edge information
struct edgeInfo {
    int src, dest;
};

// structure for update information
struct updateInfo {
    char type;
    int src, dest;
};

// HOST FUNCTIONS:
bool compareTwoEdges(const edgeInfo &a, const edgeInfo &b);
unsigned long long TC_GPU(int numVertices, int numEdges, 
                          int *csrOffsets, int *csrCords, bool isDirected);
void checkCudaError();
long long int processUpdates(updateInfo *updates, int batchSize, int numVertices, bool isDirected, 
                             int numEdges, int *csrOffsets, int *csrCords, 
                             int &numEdgesDiffCsr, int *&diffCsrOffsets, int *&diffCsrCords);

// DEVICE FUNCTIONS:
__global__ void tc_kernel(int numVertices, int *csrOffsets_d, int *csrCords_d, 
                          unsigned long long int *Tcount_d, bool isDirected);
template <typename T>
__global__ void init_kernel(T *array, T val, int arraySize);
__global__ void mark_del_modified(updateInfo *updates_d, int batchSize, 
                                  int *csrOffsets_d, int *csrCords_d, bool *modifiedCsr_d, 
                                  int *diffCsrOffsets_d, int *diffCsrCords_d, bool *modifiedDiffCsr_d);
__global__ void count_deleted_triangles(updateInfo *updates_d, int batchSize, unsigned long long int *count1_d,
                                        unsigned long long int *count2_d, unsigned long long int *count3_d, 
                                        int *csrOffsets_d, int *csrCords_d, bool *modifiedCsr_d,
                                        int *diffCsrOffsets_d, int *diffCsrCords_d, bool *modifiedDiffCsr_d);
__global__ void del_edges(int *csrCords_d, bool *modifiedCsr_d, int numEdges, 
                          int *diffCsrCords_d, bool *modifiedDiffCsr_d, int numEdgesDiffCsr);
__global__ void add_edges_csr(updateInfo *updates_d, int batchSize, int *csrOffsets_d, int *csrCords_d, 
                              bool *modifiedCsr_d, int *diffCsrOffsets_next_d);
__global__ void add_diff_edge_counts(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                     int *diffCsrOffsets_next_d);
template <typename T>
__global__ void copy(T *destArr, T *srcArr, int arraySize);
__global__ void prefix_sum(int numVertices, int off, int *diffCsrOffsets_next_d, int *tempArr_d);
__global__ void copy_edges_diffcsr(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                   int *diffCsrOffsets_next_d, int *diffCsrCords_next_d);
__global__ void add_edges_diffcsr(updateInfo *updates_d, int batchSize, int *diffCsrOffsets_d, 
                                  int *diffCsrCords_d, bool *modifiedDiffCsr_d);
__global__ void count_added_triangles(updateInfo *updates_d, int batchSize, unsigned long long int *count1_d,
                                      unsigned long long int *count2_d, unsigned long long int *count3_d, 
                                      int *csrOffsets_d, int *csrCords_d, bool *modifiedCsr_d,
                                      int *diffCsrOffsets_d, int *diffCsrCords_d, bool *modifiedDiffCsr_d);


// main function
// usage: ./a.out inputFile updateFile [percentUpdate] [isDirected]
int main(int argc, char **argv) {
    // if input or update file names are not provided then exit
    if(argc < 3) {
        printf("Enter the input and update file path in the command line.\n");
        return 0;
    }

    // read file names
    char *inputFile = argv[1];
    char *updateFile = argv[2];

    // open input and update files
    FILE *inputFilePtr = fopen(inputFile, "r");
    FILE *updateFilePtr = fopen(updateFile, "r");
    
    // if not able to open the input file then exit
    if(inputFilePtr == NULL) {
        printf("Failed to open the input file.\n");
        return 0;
    }

    // if not able to open the update file then exit
    if(updateFilePtr == NULL) {
        printf("Failed to open the update file.\n");
        return 0;
    }

    // declaration of variables
    int numVertices, numEdges;
    fscanf(inputFilePtr, "%d", &numVertices);
    fscanf(inputFilePtr, "%d", &numEdges);

    // to store the input graph in COO format
    std::vector<edgeInfo> COO(numEdges);

    // read from the input file and populate the COO
    for(int i=0; i<numEdges; i++) {
        int src, dest;
        fscanf(inputFilePtr, "%d %d", &src, &dest);

        COO[i].src = src;
        COO[i].dest = dest;
    }

    // close input file
    fclose(inputFilePtr);

    // sort the COO
    std::sort(COO.begin(), COO.end(), compareTwoEdges);

    // converting the graph in COO format to CSR format
    int *csrOffsets = (int*)malloc(sizeof(int)*(numVertices+1));
    int *csrCords = (int*)malloc(sizeof(int)*(numEdges));

    // initialize the Offsets array
    for(int i=0; i<=numVertices; i++) csrOffsets[i] = 0;

    // update the Coordinates array
    for(int i=0; i<numEdges; i++) csrCords[i] = COO[i].dest;

    // update the Offsets array
    for(int i=0; i<numEdges; i++) csrOffsets[COO[i].src+1]++;		    // store the frequency
    for(int i=0; i<numVertices; i++) csrOffsets[i+1] += csrOffsets[i];	// do cumulative sum

    std::vector<edgeInfo>().swap(COO);

    // converting the graph to CSR done

    // get graph type
    bool isDirected = true;
    if(argc >= 5 && atoi(argv[4])==0) isDirected = false;

    // compute the number of triangles
    unsigned long long gpuTC = TC_GPU(numVertices, numEdges, csrOffsets, csrCords, isDirected);
    printf("Initial graph: TC=%llu\n", gpuTC);
    
    // will be using Diff-CSR along with CSR for dynamic graph
    int numEdgesDiffCsr = 0;
    int *diffCsrOffsets, *diffCsrCords;
    diffCsrOffsets = (int*)malloc(sizeof(int)*(numVertices+1));
    diffCsrCords = NULL; 
    for(int i=0; i<=numVertices; i++)
        diffCsrOffsets[i] = 0;
    
    // for measuring time for each update
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

    // start updates
    int numUpdates, batchSize, percentUpdate;
    char type;
    int u, v;
    double totalTime = 0.0;
    
    percentUpdate = 10;
    numUpdates = percentUpdate * 0.01 * numEdges;
    if(argc >= 4) {
        percentUpdate = atoi(argv[3]);
        numUpdates = percentUpdate * 0.01 * numEdges;
    }
    if(!isDirected && numUpdates&1) numUpdates++;
    batchSize = numUpdates;
    updateInfo *updates = (updateInfo*) malloc(sizeof(updateInfo) * batchSize);
    for(int i=0; i < numUpdates; i+=batchSize) {
        for(int j=0; j<batchSize; j++) {
            fscanf(updateFilePtr, " %c", &type);
            fscanf(updateFilePtr, "%d", &u);
            fscanf(updateFilePtr, "%d", &v);

            updates[j] = {type, u, v};
            if(!isDirected) updates[++j] = {type, v, u};
        }

        // call the function to process the updates for current batch
        start = std::chrono::high_resolution_clock::now();
        gpuTC += processUpdates(updates, batchSize, numVertices, isDirected, 
                                numEdges, csrOffsets, csrCords, 
                                numEdgesDiffCsr, diffCsrOffsets, diffCsrCords);
        end = std::chrono::high_resolution_clock::now();

        // for measuring total time taken for updates
        std::chrono::duration<double, std::milli> timeTaken = end-start;
        totalTime += timeTaken.count();
        printf("Total time taken for %d percent updates: %.3f ms\n", percentUpdate, totalTime);
    }
    
    printf("Final graph: TC=%llu\n\n", gpuTC);

    // free memory allocated on host
    free(csrOffsets);
    free(csrCords);
    free(diffCsrOffsets);
    free(diffCsrCords);
    free(updates);

    // close update file
    fclose(updateFilePtr);

    return 0;
}

// comparator function
bool compareTwoEdges(const edgeInfo &a, const edgeInfo &b) {
    if(a.src != b.src) return a.src < b.src;
    return a.dest < b.dest;
}

// Triangle Counting using GPU
unsigned long long TC_GPU(int numVertices, int numEdges, 
                          int *csrOffsets, int *csrCords, bool isDirected) {
    // launch config
    const int numThreads = 1024;
    const int numBlocksV = (numVertices+numThreads-1)/numThreads;

    // variables on host
    unsigned long long int Tcount = 0;
    
    // pointers for arrays on device
    int *csrOffsets_d, *csrCords_d;
    unsigned long long int *Tcount_d;

    // allocate memory on device
    hipMalloc(&csrOffsets_d, sizeof(int)*(numVertices+1));
    hipMalloc(&csrCords_d, sizeof(int)*(numEdges));
    hipMalloc(&Tcount_d, sizeof(unsigned long long int));

    // for recording the total time taken
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy to device
    hipMemcpy(csrOffsets_d, csrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(csrCords_d, csrCords, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(Tcount_d, &Tcount, sizeof(unsigned long long int), hipMemcpyHostToDevice);

    // call kernel to compute the triangle counts
    tc_kernel<<<numBlocksV, numThreads>>>(numVertices, csrOffsets_d, csrCords_d, Tcount_d, isDirected);

    // check for error
    checkCudaError();
    
    // copy distances back to host
    hipMemcpy(&Tcount, Tcount_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    // print time taken
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Initial graph, TC using GPU - time taken: %.3f ms \n", milliseconds);

    // free up the memory
    hipFree(csrOffsets_d);
    hipFree(csrCords_d);
    hipFree(Tcount_d);
    
    return Tcount;
}

// check for cudaError
void checkCudaError() {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}

// process updates (in batches)
long long int processUpdates(updateInfo *updates, int batchSize, int numVertices, bool isDirected,
                             int numEdges, int *csrOffsets, int *csrCords,
                             int &numEdgesDiffCsr, int *&diffCsrOffsets, int *&diffCsrCords) {
    // launch config
    const int numThreads = 1024;
    const int numBlocksB = (batchSize+numThreads-1)/numThreads;
    const int numBlocksV = (numVertices+numThreads-1)/numThreads;

    // number of triangles added or deleted as a result of this batch update
    long long int Tcount;
    unsigned long long int count1, count2, count3;
    Tcount = count1 = count2 = count3 = 0;

    // pointers for arrays on device
    updateInfo *updates_d;
    unsigned long long int *count1_d, *count2_d, *count3_d;
    int *csrOffsets_d, *csrCords_d;
    int *diffCsrOffsets_d, *diffCsrCords_d;
    bool *modifiedCsr_d, *modifiedDiffCsr_d;

    // allocate memory on device
    hipMalloc(&updates_d, sizeof(updateInfo)*batchSize);
    hipMalloc(&count1_d, sizeof(unsigned long long int));
    hipMalloc(&count2_d, sizeof(unsigned long long int));
    hipMalloc(&count3_d, sizeof(unsigned long long int));
    hipMalloc(&csrOffsets_d, sizeof(int)*(numVertices+1));
    hipMalloc(&csrCords_d, sizeof(int)*(numEdges));
    hipMalloc(&diffCsrOffsets_d, sizeof(int)*(numVertices+1));
    hipMalloc(&diffCsrCords_d, sizeof(int)*(numEdgesDiffCsr));
    hipMalloc(&modifiedCsr_d, sizeof(bool)*(numEdges));
    hipMalloc(&modifiedDiffCsr_d, sizeof(bool)*(numEdgesDiffCsr));

    // for recording the time taken
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy to device
    hipMemcpy(updates_d, updates, sizeof(updateInfo)*(batchSize), hipMemcpyHostToDevice);
    hipMemcpy(count1_d, &count1, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(count2_d, &count2, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(count3_d, &count3, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(csrOffsets_d, csrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(csrCords_d, csrCords, sizeof(int)*(numEdges), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrOffsets_d, diffCsrOffsets, sizeof(int)*(numVertices+1), hipMemcpyHostToDevice);
    hipMemcpy(diffCsrCords_d, diffCsrCords, sizeof(int)*(numEdgesDiffCsr), hipMemcpyHostToDevice);
    init_kernel<bool><<<numBlocksV, numThreads>>>(modifiedCsr_d, false, numEdges);
    init_kernel<bool><<<numBlocksV, numThreads>>>(modifiedDiffCsr_d, false, numEdgesDiffCsr);

    // mark modified edges
    mark_del_modified<<<numBlocksB, numThreads>>>(updates_d, batchSize,
                                                  csrOffsets_d, csrCords_d, modifiedCsr_d,
                                                  diffCsrOffsets_d, diffCsrCords_d, modifiedDiffCsr_d);
    
    // count the number of deleted triangles
    count_deleted_triangles<<<numBlocksB, numThreads>>>(updates_d, batchSize, count1_d, count2_d, count3_d,
                                                        csrOffsets_d, csrCords_d, modifiedCsr_d,
                                                        diffCsrOffsets_d, diffCsrCords_d, modifiedDiffCsr_d);

    // kernel to delete the edges
    del_edges<<<numBlocksV, numThreads>>>(csrCords_d, modifiedCsr_d, numEdges, 
                                          diffCsrCords_d, modifiedDiffCsr_d, numEdgesDiffCsr);

    hipMemcpy(&count1, count1_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&count2, count2_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&count3, count3_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    if(isDirected) Tcount -= (count1 + count2/2 + count3/3);
    else Tcount -= (count1 + count2/2 + count3/3)/2;
    count1 = count2 = count3 = 0;

    hipMemcpy(count1_d, &count1, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(count2_d, &count2, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(count3_d, &count3, sizeof(unsigned long long int), hipMemcpyHostToDevice);

    // new diffCSR
    int *diffCsrOffsets_next_d, *diffCsrCords_next_d;
    hipMalloc(&diffCsrOffsets_next_d, sizeof(int)*(numVertices+1));
    init_kernel<int><<<numBlocksV+1, numThreads>>>(diffCsrOffsets_next_d, 0, numVertices+1);
    init_kernel<bool><<<numBlocksV, numThreads>>>(modifiedCsr_d, false, numEdges);
    
    // add edges to available space in CSR
    add_edges_csr<<<numBlocksB, numThreads>>>(updates_d, batchSize, csrOffsets_d, csrCords_d, 
                                              modifiedCsr_d, diffCsrOffsets_next_d);
    
    // get the count of valid edges in old diffCSR (for computing offsets for new diffCSR)
    add_diff_edge_counts<<<numBlocksV, numThreads>>>(numVertices, diffCsrOffsets_d, diffCsrCords_d, 
                                                     diffCsrOffsets_next_d);

    // compute offsets
    int *tempArr_d;
    hipMalloc(&tempArr_d, sizeof(int)*(numVertices+1));
    copy<int><<<numBlocksV+1, numThreads>>>(tempArr_d, diffCsrOffsets_next_d, numVertices+1);
    
    for(int off=1; off<=numVertices; off*=2) {
        prefix_sum<<<numBlocksV+1, numThreads>>>(numVertices, off, diffCsrOffsets_next_d, tempArr_d);
        copy<int><<<numBlocksV+1, numThreads>>>(tempArr_d, diffCsrOffsets_next_d, numVertices+1);
    }
    hipFree(tempArr_d);
    
    // populate new diffCSR
    hipMemcpy(&numEdgesDiffCsr, &diffCsrOffsets_next_d[numVertices], sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&diffCsrCords_next_d, sizeof(int)*(numEdgesDiffCsr));
    init_kernel<int><<<numBlocksV+1, numThreads>>>(diffCsrCords_next_d, MAX_INT, numEdgesDiffCsr);
    
    // copy edges from previous diffCSR to new diffCSr
    copy_edges_diffcsr<<<numBlocksV, numThreads>>>(numVertices, diffCsrOffsets_d, diffCsrCords_d, 
                                                   diffCsrOffsets_next_d, diffCsrCords_next_d);

    hipFree(diffCsrOffsets_d); diffCsrOffsets_d = diffCsrOffsets_next_d;
    hipFree(diffCsrCords_d); diffCsrCords_d = diffCsrCords_next_d;

    hipFree(modifiedDiffCsr_d);
    hipMalloc(&modifiedDiffCsr_d, sizeof(bool)*(numEdgesDiffCsr));
    init_kernel<bool><<<numBlocksV, numThreads>>>(modifiedDiffCsr_d, false, numEdgesDiffCsr);

    // add remaining edges to new diffCSR
    add_edges_diffcsr<<<numBlocksB, numThreads>>>(updates_d, batchSize, diffCsrOffsets_d, diffCsrCords_d, 
                                                  modifiedDiffCsr_d);

    // count the number of added triangles
    count_added_triangles<<<numBlocksB, numThreads>>>(updates_d, batchSize, count1_d, count2_d, count3_d, 
                                                      csrOffsets_d, csrCords_d, modifiedCsr_d,
                                                      diffCsrOffsets_d, diffCsrCords_d, modifiedDiffCsr_d);

    hipMemcpy(&count1, count1_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&count2, count2_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&count3, count3_d, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    if(isDirected) Tcount += (count1 + count2/2 + count3/3);
    else Tcount += (count1 + count2/2 + count3/3)/2;
    
    // copy the arrays back to the host
    hipMemcpy(csrOffsets, csrOffsets_d, sizeof(int)*(numVertices+1), hipMemcpyDeviceToHost);
    hipMemcpy(csrCords, csrCords_d, sizeof(int)*(numEdges), hipMemcpyDeviceToHost);
    hipMemcpy(diffCsrOffsets, diffCsrOffsets_d, sizeof(int)*(numVertices+1), hipMemcpyDeviceToHost);
    diffCsrCords = (int*)realloc(diffCsrCords, sizeof(int)*(numEdgesDiffCsr));
    hipMemcpy(diffCsrCords, diffCsrCords_d, sizeof(int)*(numEdgesDiffCsr), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaError();
    
    // print the time taken
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Final graph, TC using GPU - time taken: %.3f ms \n", milliseconds);

    // free up the memory
    hipFree(updates_d);
    hipFree(count1_d);
    hipFree(count2_d);
    hipFree(count3_d);
    hipFree(csrOffsets_d);
    hipFree(csrCords_d);
    hipFree(diffCsrOffsets_d);
    hipFree(diffCsrCords_d);
    hipFree(modifiedCsr_d);
    hipFree(modifiedDiffCsr_d);

    return Tcount;
}

// kernel for computing TC
// edges are u->v, v->w & w->u
__global__ void tc_kernel(int numVertices, int *csrOffsets_d, int *csrCords_d, 
                          unsigned long long int *Tcount_d, bool isDirected) {
    unsigned int u = blockDim.x*blockIdx.x + threadIdx.x;
    if(u < numVertices) {
        for(int idx1=csrOffsets_d[u]; idx1<csrOffsets_d[u+1]; idx1++) {
            int v = csrCords_d[idx1];
            if(u < v) {
                for(int idx2=csrOffsets_d[v]; idx2<csrOffsets_d[v+1]; idx2++) {
                    int w = csrCords_d[idx2];
                    if(v < w) {                
                        // check for w->u edge
                        for(int idx3=csrOffsets_d[w]; idx3<csrOffsets_d[w+1]; idx3++) {
                            if(csrCords_d[idx3]==u) atomicAdd(Tcount_d, 1);
                        }
                    }
                }
            }
        }
    }
    if(isDirected && u < numVertices) {
        for(int idx1=csrOffsets_d[u]; idx1<csrOffsets_d[u+1]; idx1++) {
            int v = csrCords_d[idx1];
            if(u > v) {
                for(int idx2=csrOffsets_d[v]; idx2<csrOffsets_d[v+1]; idx2++) {
                    int w = csrCords_d[idx2];
                    if(v > w) {                
                        // check for w->u edge
                        for(int idx3=csrOffsets_d[w]; idx3<csrOffsets_d[w+1]; idx3++) {
                            if(csrCords_d[idx3]==u) atomicAdd(Tcount_d, 1);
                        }
                    }
                }
            }
        }
    }
}

// kernel for value initialization
template <typename T>
__global__ void init_kernel(T *array, T val, int arraySize) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int numThreads = blockDim.x*gridDim.x;
    if(id < arraySize) {
        while(id < arraySize) {
            array[id] = val;
            id += numThreads;
        }
    }
}

// kernel for marking modified (to be deleted) edges in the graph (CSR and diffCSR)
__global__ void mark_del_modified(updateInfo *updates_d, int batchSize, 
                                  int *csrOffsets_d, int *csrCords_d, bool *modifiedCsr_d, 
                                  int *diffCsrOffsets_d, int *diffCsrCords_d, bool *modifiedDiffCsr_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type=='d') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        for(int i=csrOffsets_d[u]; i<csrOffsets_d[u+1]; i++) {
            if(csrCords_d[i]==v) {
                modifiedCsr_d[i] = true;
                return;
            }
        }
        for(int i=diffCsrOffsets_d[u]; i<diffCsrOffsets_d[u+1]; i++) {
            if(diffCsrCords_d[i]==v) {
                modifiedDiffCsr_d[i] = true;
                return;
            }
        }
    }
}

// kernel for counting the number of deleted triangles
__global__ void count_deleted_triangles(updateInfo *updates_d, int batchSize, unsigned long long int *count1_d,
                                        unsigned long long int *count2_d, unsigned long long int *count3_d, 
                                        int *csrOffsets_d, int *csrCords_d, bool *modifiedCsr_d,
                                        int *diffCsrOffsets_d, int *diffCsrCords_d, bool *modifiedDiffCsr_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type=='d') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        if(u!=v) {
            int count = 1;
            for(int idx1=csrOffsets_d[v]; idx1<csrOffsets_d[v+1]; idx1++) {
                int w = csrCords_d[idx1];
                if(w==MAX_INT || w==u || w==v) continue;
                if(modifiedCsr_d[idx1]) count++;
                for(int idx2=csrOffsets_d[w]; idx2<csrOffsets_d[w+1]; idx2++) {
                    if(csrCords_d[idx2] == u) {
                        if(modifiedCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedCsr_d[idx2]) count--;
                    }
                }
                for(int idx2=diffCsrOffsets_d[w]; idx2<diffCsrOffsets_d[w+1]; idx2++) {
                    if(diffCsrCords_d[idx2] == u) {
                        if(modifiedDiffCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedDiffCsr_d[idx2]) count--;
                    }
                }
                if(modifiedCsr_d[idx1]) count--;
            }
            for(int idx1=diffCsrOffsets_d[v]; idx1<diffCsrOffsets_d[v+1]; idx1++) {
                int w = diffCsrCords_d[idx1];
                if(w==MAX_INT || w==u || w==v) continue;
                if(modifiedDiffCsr_d[idx1]) count++;
                for(int idx2=csrOffsets_d[w]; idx2<csrOffsets_d[w+1]; idx2++) {
                    if(csrCords_d[idx2] == u) {
                        if(modifiedCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedCsr_d[idx2]) count--;
                    }
                }
                for(int idx2=diffCsrOffsets_d[w]; idx2<diffCsrOffsets_d[w+1]; idx2++) {
                    if(diffCsrCords_d[idx2] == u) {
                        if(modifiedDiffCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedDiffCsr_d[idx2]) count--;
                    }
                }
                if(modifiedDiffCsr_d[idx1]) count--;
            }
        }
    }
}

// kernel to delete edges
__global__ void del_edges(int *csrCords_d, bool *modifiedCsr_d, int numEdges, 
                          int *diffCsrCords_d, bool *modifiedDiffCsr_d, int numEdgesDiffCsr) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int numThreads = blockDim.x*gridDim.x;
    if(id < numEdges) {
        while(id < numEdges) {
            if(modifiedCsr_d[id]) csrCords_d[id] = MAX_INT;
            id += numThreads;
        }
    }
    id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < numEdgesDiffCsr) {
        while(id < numEdgesDiffCsr) {
            if(modifiedDiffCsr_d[id]) diffCsrCords_d[id] = MAX_INT;
            id += numThreads;
        }
    }
}

// kernel for adding edges in to the CSR and marking modified nodes
__global__ void add_edges_csr(updateInfo *updates_d, int batchSize, int *csrOffsets_d, int *csrCords_d, 
                              bool *modifiedCsr_d, int *diffCsrOffsets_next_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type=='a') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        int flag=0;
        for(int i=csrOffsets_d[u]; i<csrOffsets_d[u+1]; i++) {
            if(csrCords_d[i]==MAX_INT) {
                flag = atomicCAS(&csrCords_d[i], MAX_INT, v);
                if(flag==MAX_INT) {
                    updates_d[id].type='x';
                    modifiedCsr_d[i] = true;
                    break;
                }
            }
        }

        if(flag!=MAX_INT) atomicAdd(&diffCsrOffsets_next_d[u+1], 1);
    }
}

// kernel for counting valid edges in diffCSR (for updating offset values)
__global__ void add_diff_edge_counts(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                     int *diffCsrOffsets_next_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices) {
        for(int i=diffCsrOffsets_d[id]; i<diffCsrOffsets_d[id+1]; i++) {
            if(diffCsrCords_d[i]!=MAX_INT) diffCsrOffsets_next_d[id+1] += 1;
        }
    }
}

// kernel for copying values from one array to another array
template <typename T>
__global__ void copy(T *destArr, T *srcArr, int arraySize) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < arraySize) destArr[id] = srcArr[id];
}

// kernel for prefix sum
__global__ void prefix_sum(int numVertices, int off, int *diffCsrOffsets_next_d, int *tempArr_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if((id <= numVertices) && (id >= off)) {
        diffCsrOffsets_next_d[id] += tempArr_d[id-off];
    }
}

// kernel for copying the edges from old diffCSR to new diffCSR arrays
__global__ void copy_edges_diffcsr(int numVertices, int *diffCsrOffsets_d, int *diffCsrCords_d, 
                                   int *diffCsrOffsets_next_d, int *diffCsrCords_next_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id<numVertices) {
        int j=diffCsrOffsets_next_d[id];
        for(int i=diffCsrOffsets_d[id]; i<diffCsrOffsets_d[id+1]; i++) {
            if(diffCsrCords_d[i]!=MAX_INT) {
                diffCsrCords_next_d[j] = diffCsrCords_d[i];
                j++;
            }
        }
    }
}

// kernel for adding remaining edges (from udpates) to diffCSR
__global__ void add_edges_diffcsr(updateInfo *updates_d, int batchSize, int *diffCsrOffsets_d, 
                                  int *diffCsrCords_d, bool *modifiedDiffCsr_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type=='a') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        for(int i=diffCsrOffsets_d[u]; i<diffCsrOffsets_d[u+1]; i++) {
            if(diffCsrCords_d[i]==MAX_INT) {
                if(atomicCAS(&diffCsrCords_d[i], MAX_INT, v)==MAX_INT) {
                    updates_d[id].type='y';
                    modifiedDiffCsr_d[i] = true;
                    break;
                }
            }
        }
    }
}

// kernel for counting the number of newly formed triangles
__global__ void count_added_triangles(updateInfo *updates_d, int batchSize, unsigned long long int *count1_d,
                                      unsigned long long int *count2_d, unsigned long long int *count3_d, 
                                      int *csrOffsets_d, int *csrCords_d, bool *modifiedCsr_d,
                                      int *diffCsrOffsets_d, int *diffCsrCords_d, bool *modifiedDiffCsr_d) {
    unsigned int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < batchSize && updates_d[id].type!='d') {
        int u = updates_d[id].src;
        int v = updates_d[id].dest;
        if(u!=v) {
            int count = 1;
            for(int idx1=csrOffsets_d[v]; idx1<csrOffsets_d[v+1]; idx1++) {
                int w = csrCords_d[idx1];
                if(w==MAX_INT || w==u || w==v) continue;
                if(modifiedCsr_d[idx1]) count++;
                for(int idx2=csrOffsets_d[w]; idx2<csrOffsets_d[w+1]; idx2++) {
                    if(csrCords_d[idx2] == u) {
                        if(modifiedCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedCsr_d[idx2]) count--;
                    }
                }
                for(int idx2=diffCsrOffsets_d[w]; idx2<diffCsrOffsets_d[w+1]; idx2++) {
                    if(diffCsrCords_d[idx2] == u) {
                        if(modifiedDiffCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedDiffCsr_d[idx2]) count--;
                    }
                }
                if(modifiedCsr_d[idx1]) count--;
            }
            for(int idx1=diffCsrOffsets_d[v]; idx1<diffCsrOffsets_d[v+1]; idx1++) {
                int w = diffCsrCords_d[idx1];
                if(w==MAX_INT || w==u || w==v) continue;
                if(modifiedDiffCsr_d[idx1]) count++;
                for(int idx2=csrOffsets_d[w]; idx2<csrOffsets_d[w+1]; idx2++) {
                    if(csrCords_d[idx2] == u) {
                        if(modifiedCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedCsr_d[idx2]) count--;
                    }
                }
                for(int idx2=diffCsrOffsets_d[w]; idx2<diffCsrOffsets_d[w+1]; idx2++) {
                    if(diffCsrCords_d[idx2] == u) {
                        if(modifiedDiffCsr_d[idx2]) count++;
                        if(count==1)      atomicAdd(count1_d, 1);
                        else if(count==2) atomicAdd(count2_d, 1);
                        else if(count==3) atomicAdd(count3_d, 1);
                        if(modifiedDiffCsr_d[idx2]) count--;
                    }
                }
                if(modifiedDiffCsr_d[idx1]) count--;
            }
        }
    }
}